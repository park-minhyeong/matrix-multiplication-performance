#include "hip/hip_runtime.h"
#include "../include/matrix.h"
#include <hip/hip_runtime.h>

// Naive CUDA kernel (global memory only)
__global__ void naive_matmul_kernel(const float* A, const float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; ++k)
            sum += A[row * N + k] * B[k * N + col];
        C[row * N + col] = sum;
    }
}

// Host wrapper for naive CUDA kernel
void naive_cuda_matmul(const Matrix& A, const Matrix& B, Matrix& C) {
    int N = A.N;
    size_t bytes = N * N * sizeof(float);
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);
    hipMemcpy(d_A, A.data, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data, bytes, hipMemcpyHostToDevice);
    dim3 block(16, 16);
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);
    naive_matmul_kernel<<<grid, block>>>(d_A, d_B, d_C, N);
    hipMemcpy(C.data, d_C, bytes, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
